#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <thread>

void mb::UpdateMath(IterationT addIt){
    std::thread *ArrThreads[NThreads];
    for(unsigned long L, R, i = 0; i < NThreads; ++i){
        L =  i   *N/NThreads;
        R = (i+1)*N/NThreads;
        ArrThreads[i] = new std::thread(&mb::UpdateMathLim, this, L, R, addIt);
    }
    for(unsigned long i = 0; i < NThreads; ++i){
        ArrThreads[i]->join();
    }
    numIt += addIt;
}

void mb::UpdateMathLim(IterationT addIt){
    unsigned long i = threadIdx.x;
    unsigned long NThreads = blockDim.x;
    unsigned long N = GetSize().x*GetSize().y;
    unsigned long L =  i   *N/NThreads;
    unsigned long R = (i+1)*N/NThreads;
    for(unsigned long i = L; i < R; ++i){
        if(!Check[i]) continue;
        IterationT it;
        ComplexNum z = Z[i], c = C[i];
        for(it = 0; it < addIt; ++it){
            z = z*z + c;
            if(std::norm(z) > bailout_sqr){
                Z[i] = z; IT[i] += it;
                Check[i] = false;
                UpdatePixel(i);
                break;
            }
        }
        if(Check[i]){
            Z[i] = z;
            IT[i] += it;
        }
    }
}
